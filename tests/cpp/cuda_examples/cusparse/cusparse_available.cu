#include <hip/hip_runtime.h>

#include <hipsparse.h>

#include <iostream>

// A simple program to make sure CUSPARSE is working from C.

// Compile with: nvcc cusparse_available.cu -o cusparse -lcusparse

int main(int argc, char** argv) {

	hipsparseHandle_t context = 0;
	std::cout << "Making context..." << std::endl;

	hipsparseStatus_t status = hipsparseCreate(&context);

	std::cout << "Status: " << status << std::endl;

	if(status != HIPSPARSE_STATUS_SUCCESS) {

		std::cout << "Failed!" << std::endl;

	}

	

	int cusparseVersion = 0;
    int cudartVersion = 0;
    int cudadriVersion = 0;

	std::cout << "Getting version..." << std::endl;

	status = hipsparseGetVersion(context, &cusparseVersion);
    hipRuntimeGetVersion(&cudartVersion);
    hipDriverGetVersion(&cudadriVersion);

	std::cout << "cusparse Version: " << cusparseVersion << std::endl;
    std::cout << "cudart Version: " << cudartVersion << std::endl;
    std::cout << "cudadriver Version: " << cudadriVersion << std::endl;

	std::cout << "Status: " << status << std::endl;

	if(status != HIPSPARSE_STATUS_SUCCESS) {

		std::cout << "Failed!" << std::endl;

	}

	

	std::cout << "Destroying context..." << std::endl;

	status = hipsparseDestroy(context);

	std::cout << "Status: " << status << std::endl;

	if(status != HIPSPARSE_STATUS_SUCCESS) {

		std::cout << "Failed!" << std::endl;

	}

}